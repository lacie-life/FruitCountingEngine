#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include <iostream>

__global__ void _cuda_feature2bbox(VPIKeypoint *kpts, VPIKLTTrackedBoundingBox *bboxes,
                                   VPIHomographyTransform2D *preds, uint32_t size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if( index >= size ) return;

    VPIKeypoint kpt = kpts[index];
    VPIKLTTrackedBoundingBox *track = &bboxes[index];
    VPIHomographyTransform2D *xform = &preds[index];

    // bbox
    memset(track, 0, sizeof(VPIKLTTrackedBoundingBox));
    track->bbox.xform.mat3[0][0] = 1;
    track->bbox.xform.mat3[1][1] = 1;
    track->bbox.xform.mat3[0][2] = float(kpt.x) - 15.5f;
    track->bbox.xform.mat3[1][2] = float(kpt.y) - 15.5f;
    track->bbox.xform.mat3[2][2] = 1;

    track->bbox.width     = 32.f;
    track->bbox.height    = 32.f;
    track->trackingStatus = 0;
    track->templateStatus = 1;

    // pred
    memset(xform, 0, sizeof(VPIHomographyTransform2D));
    xform->mat3[0][0] = 1;
    xform->mat3[1][1] = 1;
    xform->mat3[2][2] = 1;
}

int cuda_feature2bbox(hipStream_t &stream, void *kpts, void *input_box, void *input_pred, uint32_t size)
{
    _cuda_feature2bbox<<< (size+THREAD-1)/THREAD, THREAD, 0, stream >>>( (VPIKeypoint*)kpts, (VPIKLTTrackedBoundingBox*)input_box,
                                                                         (VPIHomographyTransform2D*)input_pred, size);
    hipStreamSynchronize(stream);
    return 0;
}
