#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaDraw.h"
#include "cudaAlphaBlend.cuh"


// TODO for rect/fill/line
//    - make versions that only accept image (as both input/output)
//    - add line width/line color
//    - add overloads for single shape/multiple shapes
//    - benchmarking of copy vs alternate kernel when input != output
//    - overloads using int2 for coordinates
//    - add a template parameter for alpha blending

#define MIN(a,b)  (a < b ? a : b)
#define MAX(a,b)  (a > b ? a : b)

template<typename T> inline __device__ __host__ T sqr(T x) 				    { return x*x; }

inline __device__ __host__ float dist2(float x1, float y1, float x2, float y2) { return sqr(x1-x2) + sqr(y1-y2); }
inline __device__ __host__ float dist(float x1, float y1, float x2, float y2)  { return sqrtf(dist2(x1,y1,x2,y2)); }


//----------------------------------------------------------------------------
// Circle drawing (find if the distance to the circle <= radius)
//----------------------------------------------------------------------------						 
template<typename T>
__global__ void gpuDrawCircle( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int cx, int cy, float radius2, const float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int dx = x - cx;
	const int dy = y - cy;
	
	// if x,y is in the circle draw it
	if( dx * dx + dy * dy < radius2 ) 
	{
		const int idx = y * imgWidth + x;
		img[idx] = cudaAlphaBlend(img[idx], color);
	}
}

// cudaDrawCircle
hipError_t cudaDrawCircle( void* input, void* output, size_t width, size_t height, imageFormat format, int cx, int cy, float radius, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 || radius <= 0 )
		return hipErrorInvalidValue;

	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the circle
	const int diameter = ceilf(radius * 2.0f);
	const int offset_x = cx - radius;
	const int offset_y = cy - radius;
	
	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(diameter,blockDim.x), iDivUp(diameter,blockDim.y));

	#define LAUNCH_DRAW_CIRCLE(type) \
		gpuDrawCircle<type><<<gridDim, blockDim>>>((type*)output, width, height, offset_x, offset_y, cx, cy, radius*radius, color)
	
	if( format == IMAGE_RGB8 )
		LAUNCH_DRAW_CIRCLE(uchar3);
	else if( format == IMAGE_RGBA8 )
		LAUNCH_DRAW_CIRCLE(uchar4);
	else if( format == IMAGE_RGB32F )
		LAUNCH_DRAW_CIRCLE(float3); 
	else if( format == IMAGE_RGBA32F )
		LAUNCH_DRAW_CIRCLE(float4);
	else
	{
		imageFormatErrorMsg(LOG_CUDA, "cudaDrawCircle()", format);
		return hipErrorInvalidValue;
	}
		
	return hipGetLastError();
}


//----------------------------------------------------------------------------
// Line drawing (find if the distance to the line <= line_width)
// Distance from point to line segment - https://stackoverflow.com/a/1501725
//----------------------------------------------------------------------------
inline __device__ float lineDistanceSquared(float x, float y, float x1, float y1, float x2, float y2) 
{
	const float d = dist2(x1, y1, x2, y2);
	const float t = ((x-x1) * (x2-x1) + (y-y1) * (y2-y1)) / d;
	const float u = MAX(0, MIN(1, t));
	
	return dist2(x, y, x1 + u * (x2 - x1), y1 + u * (y2 - y1));
}
				 
template<typename T>
__global__ void gpuDrawLine( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int x1, int y1, int x2, int y2, const float4 color, float line_width2 ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y + offset_y;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	if( lineDistanceSquared(x, y, x1, y1, x2, y2) <= line_width2 )
	{
		const int idx = y * imgWidth + x;
		img[idx] = cudaAlphaBlend(img[idx], color);
	}
}

// cudaDrawLine
hipError_t cudaDrawLine( void* input, void* output, size_t width, size_t height, imageFormat format, int x1, int y1, int x2, int y2, const float4& color, float line_width )
{
	if( !input || !output || width == 0 || height == 0 || line_width <= 0 )
		return hipErrorInvalidValue;
	
	// check for lines < 2 pixels in length
	if( dist(x1,y1,x2,y2) < 2.0 )
	{
		LogWarning(LOG_CUDA "cudaDrawLine() - line has length < 2, skipping (%i,%i) (%i,%i)\n", x1, y1, x2, y2);
		return hipSuccess;
	}
	
	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// find a box around the line
	const int left = MIN(x1,x2) - line_width;
	const int right = MAX(x1,x2) + line_width;
	const int top = MIN(y1,y2) - line_width;
	const int bottom = MAX(y1,y2) + line_width;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(right - left, blockDim.x), iDivUp(bottom - top, blockDim.y));

	#define LAUNCH_DRAW_LINE(type) \
		gpuDrawLine<type><<<gridDim, blockDim>>>((type*)output, width, height, left, top, x1, y1, x2, y2, color, line_width * line_width)
	
	if( format == IMAGE_RGB8 )
		LAUNCH_DRAW_LINE(uchar3);
	else if( format == IMAGE_RGBA8 )
		LAUNCH_DRAW_LINE(uchar4);
	else if( format == IMAGE_RGB32F )
		LAUNCH_DRAW_LINE(float3); 
	else if( format == IMAGE_RGBA32F )
		LAUNCH_DRAW_LINE(float4);
	else
	{
		imageFormatErrorMsg(LOG_CUDA, "cudaDrawLine()", format);
		return hipErrorInvalidValue;
	}
		
	return hipGetLastError();
}



//----------------------------------------------------------------------------
// Rect drawing (a grid of threads is launched over the rect)
//----------------------------------------------------------------------------
template<typename T>
__global__ void gpuDrawRect( T* img, int imgWidth, int imgHeight, int x0, int y0, int boxWidth, int boxHeight, const float4 color ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight || x < 0 || y < 0 )
		return;

	const int idx = y * imgWidth + x;
	img[idx] = cudaAlphaBlend(img[idx], color);
}


// cudaDrawRect
hipError_t cudaDrawRect( void* input, void* output, size_t width, size_t height, imageFormat format, int left, int top, int right, int bottom, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// if the input and output images are different, copy the input to the output
	// this is because we only launch the kernel in the approximate area of the circle
	if( input != output )
		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
		
	// make sure the coordinates are ordered
	if( left > right )
	{
		const int swap = left;
		left = right;
		right = swap;
	}
	
	if( top > bottom )
	{
		const int swap = top;
		top = bottom;
		bottom = swap;
	}
	
	const int boxWidth = right - left;
	const int boxHeight = bottom - top;
	
	if( boxWidth <= 0 || boxHeight <= 0 )
	{
		LogError("cudaDrawRect() -- rect had width/height <= 0  left=%i top=%i right=%i bottom=%i\n", left, top, right, bottom);
		return hipErrorInvalidValue;
	}

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));
			
	#define LAUNCH_DRAW_RECT(type) \
		gpuDrawRect<type><<<gridDim, blockDim>>>((type*)output, width, height, left, top, boxWidth, boxHeight, color)
	
	if( format == IMAGE_RGB8 )
		LAUNCH_DRAW_RECT(uchar3);
	else if( format == IMAGE_RGBA8 )
		LAUNCH_DRAW_RECT(uchar4);
	else if( format == IMAGE_RGB32F )
		LAUNCH_DRAW_RECT(float3); 
	else if( format == IMAGE_RGBA32F )
		LAUNCH_DRAW_RECT(float4);
	else
	{
		imageFormatErrorMsg(LOG_CUDA, "cudaDrawRect()", format);
		return hipErrorInvalidValue;
	}
	
	return hipGetLastError();
}